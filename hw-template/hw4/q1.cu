
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include "vector"
#include <math.h>
#include <stdlib.h>
#include <cmath>
#include <stdio.h>
using namespace std;

__global__ void min2(int *arr, int *i) {
    int a = arr[2 * *i * blockIdx.x]; int b = arr[2 * *i * blockIdx.x + *i];
    if (a < b) {
        arr[2 * *i * blockIdx.x] = a;
    } else {
        arr[2 * *i * blockIdx.x] = b;
    }
}


int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;

            (*len)++;
            arr->push_back(atoi(s.c_str()));

        }
    }
    return 1;
}

void a(vector<int> arr, int len) {
    int full_size = len * sizeof(int);

    // Full array
    int *d_arr;

    hipMalloc((void **)&d_arr, full_size);
    
    int N = len/2;

    int i = 1;
    int *d_i;
    hipMalloc((void **)&d_i, sizeof(int));
    while (2*i < len) {
        // Copy array and i over
        hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);
        hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);

        min2<<<N,1>>>(d_arr, d_i);

        // Update
        i *= 2 ;
        N = (int)((len + 1) / (2 * i));
        hipMemcpy(arr.data(), d_arr, full_size, hipMemcpyDeviceToHost);
    }
    
    // Sequential compare
    int min;
    int a = arr[0]; int b = arr[i];
    if (a < b) {
        min = a;
    } else {
        min = b;
    }

    cout<<"Minimum: " << min << endl;

    hipFree(d_arr); hipFree(d_i);
}

int main () {
    vector<int> arr;
    int len = 0;
    if (!populate_array(&arr, &len)) {
      return 0;
    }

    a(arr, len);
    
    return 0;
}
