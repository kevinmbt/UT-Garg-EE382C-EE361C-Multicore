
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include "vector"
#include <math.h>
#include <stdlib.h>
#include <cmath>
#include <stdio.h>
using namespace std;

__global__ void min2(int *arr, int *i) {
    int a = arr[2 * *i * blockIdx.x]; int b = arr[2 * *i * blockIdx.x + *i];
    if (a < b) {
        arr[2 * *i * blockIdx.x] = a;
    } else {
        arr[2 * *i * blockIdx.x] = b;
    }
}


__global__ void remainder_array(int n, int *a, int *b){
	  int index = threadIdx.x;
		int stride = blockDim.x;
		for (int i = index; i < n; i += stride){
			b[i] = a[i]%10;
		}

}

void stream_arr_to_file(int *b, int size){
	 ofstream myfile ("q1.txt");
		  if (myfile.is_open())
			{
				for(int count = 0; count < size; count ++){
					myfile << b[count] << " , ";
				}
				myfile.close();	
			}
		else cout << "Unable to open file" << endl;
}

 


int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;

            (*len)++;
            arr->push_back(atoi(s.c_str()));

        }
    }
    return 1;
}

void a(vector<int> arr, int len) {
    int full_size = len * sizeof(int);

    // Full array
    int *d_arr;

    hipMalloc((void **)&d_arr, full_size);
    
    int N = len/2;

    int i = 1;
    int *d_i;
    hipMalloc((void **)&d_i, sizeof(int));
    while (2*i < len) {
        // Copy array and i over
        hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);
        hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);

        min2<<<N,1>>>(d_arr, d_i);

        // Update
        i *= 2 ;
        N = (int)((len + 1) / (2 * i));
        hipMemcpy(arr.data(), d_arr, full_size, hipMemcpyDeviceToHost);
    }
    
    // Sequential compare
    int min;
    int a = arr[0]; int b = arr[i];
    if (a < b) {
        min = a;
    } else {
        min = b;
    }

    cout<<"Minimum: " << min << endl;

    hipFree(d_arr); hipFree(d_i);
}

void b(vector<int> a, int len){
		int full_size = len * sizeof(int);

		int *a_arr;
		hipMalloc((void **) &a_arr, full_size);
		int *b_arr;
		hipMalloc((void **) &b_arr, full_size);

		hipMemcpy(a_arr, a.data(), full_size, hipMemcpyHostToDevice);
		hipMemcpy(b_arr, a.data(), full_size, hipMemcpyHostToDevice);

		remainder_array<<<1, 256>>> (len, a_arr, b_arr);

    hipMemcpy(a.data(), b_arr, full_size, hipMemcpyDeviceToHost);

		stream_arr_to_file(a.data(),len);

		hipFree(a_arr);
		hipFree(b_arr);
}
   	 

int main () {
    vector<int> arr;
    int len = 0;
    if (!populate_array(&arr, &len)) {
      return 0;
    }

    a(arr, len);

    if (!populate_array(&arr, &len)) {
      return 0;
    }
		b(arr, len);
    
    return 0;
}
