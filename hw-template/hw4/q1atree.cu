#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include "vector"
#include <math.h>
#include <cmath>
using namespace std;

__global__ void min2(int *arr, int i) {
    int a = arr[2 * i * blockIdx.x]; int b = arr[2 * i * blockIdx.x + i];
    if (a < b) {
        arr[2 * i * blockIdx.x] = a;
    } else {
        arr[2 * i * blockIdx.x] = b;
    }
    // if (2 * blockIdx.x + 1 >= len) {
    //     result[blockIdx.x] = arr[2 * blockIdx.x];
    // } else {
    //     a = arr[2 * blockIdx.x]; b = arr[2 * blockIdx.x + 1];
    //     if (a < b) {
    //         result[blockIdx.x] = a;
    //     } else {
    //         result[blockIdx.x] = b;
    //     }
    // }
}

int round_up(double val) {
    if (fabs(val-(int)val) < 0.00001) {
        return (int) val;
    }
    else return (int) val + 1;
}

int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;
            (*len)++;
            arr->push_back(stoi(s));
        }
    }
    return 1;
}

int main () {
    vector<int> arr;
    int len = 0;
    if (!populate_array(&arr, &len)) {
      return 0;
    }

    // int levels = round_up(log2(len));

    int full_size = len * sizeof(int);

    // Full array and result of sequential phase
    int *d_arr;

    hipMalloc((void **)&d_arr, full_size);
    hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);
    int N = len/2

    int i = 1;
    while (2*i < len) {
        min2<<<N,1>>>(d_arr, i);

        // hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

        // Update
        i *=2 ;
        N = (int)((len + 1) / (2 * i));
    }
    fin_arr = (int *)malloc(full_size);
    hipMemcpy(fin_arr, d_arr, full_size, hipMemcpyDeviceToHost);
    // Sequential comp
    int min;
    int a = arr[2 * i]; int b = arr[2 * i * blockIdx.x + i];
    if (a < b) {
        min = a;
    } else {
        min = b;
    }
    cout<<min<<endl;
    return 0;

    // for (int i = 0; i < levels; i++) {
    //     int *d_result;
    //     int *result;
    //     int n = round_up(len / pow(2, 1+i));

        
    //     hipMalloc((void **)&d_result, n * sizeof(int));

    //     result = (int *)malloc(n * sizeof(int));

    //     min2<<<n, 1>>>(d_arr, d_result, len);

    //     // Save results
    //     hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    //     // // Use results as new source
    //     arr.assign(result, result + n * sizeof(int))
    //     arr.resize(n * sizeof(int));
    //     full_size = n * sizeof(int);
        
    //     // Cleanup
    //     free(result);
    //     hipFree(d_result);
    // }
    // hipFree(d_arr); 
}