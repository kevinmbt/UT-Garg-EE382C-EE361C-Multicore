
#include <hip/hip_runtime.h>
#include<atomic>
#include <iostream>
#include "vector"
#include <fstream>
#include <sstream>
#include <string>
using namespace std;

__global__ void count_a(int *arr, int *B, int *chunk_len, int *len) {
    int start = threadIdx.x * *chunk_len;
    int i = start;
    while (i - start < *chunk_len && i < *len) {
        atomicAdd(B + (arr[i] / 100), 1);
        i++;
    }
}

__global__ void count_b(int *arr, int *B, int *chunk_len, int *len) {
    // Initialize shared array
    __shared__ int s_B[10];
    for (int i = threadIdx.x; i < 10; i += 1) {
        s_B[i] = 0;
    }
    __syncthreads();

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int start = index * *chunk_len;
    int i = start;
    while(i - start < *chunk_len && i < *len) {
        atomicAdd(s_B + (arr[i] / 100), 1);
        i++;
    }
    __syncthreads();
    if (threadIdx.x==0) {
        for (int j = 0; j < 10; j++) {
            atomicAdd(B + j, s_B[j]);
        }
    }
}

int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;

            (*len)++;
            arr->push_back(stoi(s));

        }
    }
    return 1;
}

void a(vector<int> arr, int len) {
    int size = 10 * sizeof(int);
    int full_size = len * sizeof(int);

    int B[10] = {};

    int *d_B;
    hipMalloc((void**)&d_B, size);

    int *d_arr;
    hipMalloc((void **)&d_arr, full_size);

    hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);


    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int chunk_len = (int)(len/100) + 1;
    int *d_chunk_len;
    int *d_len;
    hipMalloc((void **)&d_chunk_len, sizeof(int));
    hipMalloc((void **)&d_len, sizeof(int));
    hipMemcpy(d_chunk_len, &chunk_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);

    cout<< chunk_len<<endl;

    count_a<<<1, 100>>>(d_arr, d_B, d_chunk_len, d_len);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        cout << i << ": " << B[i] << endl;
    }
    hipFree(d_arr); hipFree(d_B); hipFree(d_chunk_len); hipFree(d_len);
}

void b(vector<int> arr, int len) {
    int size = 10 * sizeof(int);
    int full_size = len * sizeof(int);

    int B[10] = {};

    int *d_B;
    hipMalloc((void**)&d_B, size);

    int *d_arr;
    hipMalloc((void **)&d_arr, full_size);

    hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);

    // Number of threads per block
    int thds = (int) (len / 20) + 1;

    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int chunk_len = (int) len/(20 * thds) + 1;
    int *d_chunk_len;
    int *d_len;
    hipMalloc((void **)&d_chunk_len, sizeof(int));
    hipMalloc((void **)&d_len, sizeof(int));
    hipMemcpy(d_chunk_len, &chunk_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);
    
    count_b<<<20, thds>>>(d_arr, d_B, d_chunk_len, d_len);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        cout << i << ": " << B[i] << endl;
    }
    hipFree(d_arr); hipFree(d_B); hipFree(d_chunk_len); hipFree(d_len);
}

int main () {
    vector<int> arr;
    int len = 0;
    if (!populate_array(&arr, &len)) {
      return 0;
    }

    a(arr, len);
    b(arr, len);
}