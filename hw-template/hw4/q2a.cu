
#include <hip/hip_runtime.h>
#include<atomic>
#include <iostream>
#include "vector"
#include <fstream>
#include <sstream>
#include <string>
using namespace std;

__global__ void count_a(int *arr, int *B, int *chunk_len, int *len) {
    int start = threadIdx.x * *chunk_len;
    int i = start;
    while (i - start < *chunk_len && i < *len) {
        atomicAdd(B + (arr[i] / 100), 1);
        i++;
    }
}

__global__ void count_b(int *arr, int *B, int *chunk_len, int *len) {
    // Initialize shared array
    __shared__ int s_B[10];
    for (int i = threadIdx.x; i < 10; i += 1) {
        s_B[i] = 0;
    }
    __syncthreads();

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int start = index * *chunk_len;
    int i = start;
    while(i - start < *chunk_len && i < *len) {
        atomicAdd(s_B + (arr[i] / 100), 1);
        i++;
    }
    __syncthreads();
    if (threadIdx.x==0) {
        for (int j = 0; j < 10; j++) {
            atomicAdd(B + j, s_B[j]);
        }
    }
}

__global__ void sum_c(int *B, int *C) {
    if (threadIdx.x < 10) C[threadIdx.x] = B[threadIdx.x];
    __syncthreads();
    for (int d = 1; d < 10; d*=2) {
        int val;
        if (threadIdx.x >= d) {
            val = C[threadIdx.x - d];
        }
        __syncthreads();
        if (threadIdx.x >= d) {
            C[threadIdx.x] += val;
        }
        __syncthreads();
    }
}

int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;

            (*len)++;
            arr->push_back(stoi(s));

        }
    }
    return 1;
}

void a(vector<int> arr, int len) {
    int size = 10 * sizeof(int);
    int full_size = len * sizeof(int);

    int B[10] = {};

    int *d_B;
    hipMalloc((void**)&d_B, size);

    int *d_arr;
    hipMalloc((void **)&d_arr, full_size);

    hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);


    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int chunk_len = (int)(len/100) + 1;
    int *d_chunk_len;
    int *d_len;
    hipMalloc((void **)&d_chunk_len, sizeof(int));
    hipMalloc((void **)&d_len, sizeof(int));
    hipMemcpy(d_chunk_len, &chunk_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);

    cout<< chunk_len<<endl;

    count_a<<<1, 100>>>(d_arr, d_B, d_chunk_len, d_len);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        cout << i << ": " << B[i] << endl;
    }
    hipFree(d_arr); hipFree(d_B); hipFree(d_chunk_len); hipFree(d_len);
}

int* b(vector<int> arr, int len) {
    int size = 10 * sizeof(int);
    int full_size = len * sizeof(int);

    int B[10] = {};

    int *d_B;
    hipMalloc((void**)&d_B, size);

    int *d_arr;
    hipMalloc((void **)&d_arr, full_size);

    hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);

    // Number of threads per block
    int thds = (int) (len / 20) + 1;

    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int chunk_len = (int) len/(20 * thds) + 1;
    int *d_chunk_len;
    int *d_len;
    hipMalloc((void **)&d_chunk_len, sizeof(int));
    hipMalloc((void **)&d_len, sizeof(int));
    hipMemcpy(d_chunk_len, &chunk_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);
    
    count_b<<<20, thds>>>(d_arr, d_B, d_chunk_len, d_len);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        cout << i << ": " << B[i] << endl;
    }
    hipFree(d_arr); hipFree(d_B); hipFree(d_chunk_len); hipFree(d_len);

    int* retval = new int[10];
    copy(B, B+10, retval);
    return retval;
}

void c(int * B) {
    for (int i = 0; i < 10; i++) {
        cout << i << " old: " << B[i] << endl;
    }
    int *d_B; int *d_C;
    int size = 10 * sizeof(int);
    int *C = (int*)malloc(size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    sum_c<<<1,10>>>(d_B, d_C);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        cout << i << ": " << C[i] << endl;
    }

    hipFree(d_B); hipFree(d_C);
    free(C);
}

int main () {
    vector<int> arr;
    int len = 0;
    if (!populate_array(&arr, &len)) {
      return 0;
    }

    a(arr, len);
    int* B = b(arr, len);
    c(B);
    delete [] B;
}