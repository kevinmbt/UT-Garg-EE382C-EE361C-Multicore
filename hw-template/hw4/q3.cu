
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include "vector"
#include <math.h>
#include <stdlib.h>
#include <cmath>
#include <stdio.h>
using namespace std;

__global__ void odd_count(int n, int *a, int *odd_cnt){
	  int index = threadIdx.x;
		int stride = blockDim.x;
		for (int i = index; i < n; i += stride){
				if(a[i] %2 == 1){
					atomicAdd(odd_cnt, 1);
				}
		}
} 

__global__ void fill_odd_array(int n, int *a, int *b){
		int index = threadIdx.x;
		int stride = blockDim.x;
		for (int i = index; i < n; i += stride){
				if(a[i] %2 == 1){
					b[i] = a[i];
				}
		}

}

void stream_arr_to_file(int *b, int size){
	  ofstream myfile ("q3.txt");
		  if (myfile.is_open())
			{
				for(int count = 0; count < size; count ++){
					myfile << b[count] << "," ;
				}
				myfile.close();	
			}
		else cout << "Unable to open file";
}


int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;

            (*len)++;
            arr->push_back(atoi(s.c_str()));

        }
    }
    return 1;
}

void a(vector<int> a, int len){
		int full_size = len * sizeof(int);

		int *a_arr;
		hipMalloc((void **) &a_arr, full_size);
		int *odd_cnt;
		hipMallocManaged(&odd_cnt, 4);
		*odd_cnt = 0;

    int odd_cnt_arr[1] = {};

		hipMemcpy(a_arr, a.data(), full_size, hipMemcpyHostToDevice);

		odd_count<<<1, 256>>> (len, a_arr, odd_cnt);
    
    hipMemcpy(odd_cnt_arr, odd_cnt, sizeof(int), hipMemcpyDeviceToHost);

		int odd_size = odd_cnt_arr[0] * sizeof(int);

		vector<int> odd_arr; 

		int *b_arr;
		hipMalloc((void **) &b_arr, odd_size);
		hipMemcpy(a_arr, a.data(), full_size, hipMemcpyHostToDevice);
		hipMemcpy(b_arr, a.data(), odd_size, hipMemcpyHostToDevice);

		fill_odd_array<<<1, 256>>> (*odd_cnt, a_arr, b_arr);
    
		hipMemcpy(a.data(), b_arr, full_size, hipMemcpyDeviceToHost);

		stream_arr_to_file(a.data(), odd_cnt_arr[0]);

		hipFree(a_arr);
		hipFree(b_arr);
		hipFree(odd_cnt);
}
   	 

int main () {
    vector<int> arr;
    int len = 0;
    if (!populate_array(&arr, &len)) {
      return 0;
    }

    a(arr, len);
    
    return 0;
}
