
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include "vector"
using namespace std;

int populate_array(vector<string>* arr) {
  ifstream infile( "inp.txt" );
  if (!infile.is_open()) {
    cout<<"File failed to open"<<endl;
    return 0;
  }
  string line;
  while (getline(infile, line))
  {
    istringstream ss(line);

    vector <string> record;

    while (ss)
    {
      string s;
      if (!getline(ss, s, ',')) break;
      arr->push_back( s );
    }
  }
  return 1;
}

int main () {
  vector<string> arr;
  if (!populate_array(&arr)) {
    return 0;
  }
  cout<<arr[0]<<endl;
}
