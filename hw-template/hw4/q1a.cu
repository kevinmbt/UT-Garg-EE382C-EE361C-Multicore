#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include "vector"
#include <math.h>
#include <cmath>
using namespace std;

int populate_array(vector<int>* arr, int* len) {
  ifstream infile( "inp.txt" );
  if (!infile.is_open()) {
    cout<<"File failed to open"<<endl;
    return 0;
  }
  string line;
  while (getline(infile, line))
  {
    istringstream ss(line);

    while (ss)
    {
      string s;
      if (!getline(ss, s, ',')) break;
      (*len)++;
      arr->push_back(stoi(s));
    }
  }
  return 1;
}

__global__ void min_seq(int *arr, int *seq_result, int chunk_len, int full_len) {
  // TODO: Iterate over chunk until done or until N is reached
}

int main () {
  vector<int> arr;
  int len = 0;
  if (!populate_array(&arr, &len)) {
    return 0;
  }
  cout<<arr[0]<<endl;
  cout<<len<<endl;
  cout<<log(log(len))<<endl;

  int N = (int) (len / log(log(len)))

  int full_size = len * sizeof(int);
  int seq_result_size = N * sizeof(int);

  // Full array and result of sequential phase
  int *d_arr;
  int *seq_result;
  int *d_seq_result;

  hipMalloc((void **)&d_arr, full_size);
  hipMalloc((void **)&d_a_result, seq_result_size)

  seq_result = (int *)malloc(seq_result_size);

  // Copy full array
  hipMemcpy(d_arr, arr.data(), full_size, hipMemcpyHostToDevice);

  // TODO: Figure out if rounding will cause any missed elements
  min_seq<<<N, 1>>>(d_arr, d_seq_result, );

  // Save results of sequential phase
  hipMemcpy(seq_result, d_seq_result, seq_result_size, hipMemcpyDeviceToHost);

  // Cleanup
  free(seq_result);
  hipFree(d_arr); hipFree(d_seq_result);



}
